
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <iostream>
#include <vector>
#include<cmath>
#include <fstream>
using namespace std;

__global__ void compute_JST_flux(const double *__restrict u, double *__restrict dudt,
                          double uL, double uR, int n) {

    double flux_inv_p1, flux_inv_0, flux_inv, flux_vis;
    double delta_up15, delta_up05, delta_um05;

    double um_loc_m2, um_loc_m1, um_loc_0, um_loc_p1, um_loc_p2;
    double h_p05, h_m05;

    double um1, u0, up1, up2;

    // artificial viscosity
    double coeff_visc_2 = 0.5;
    double coeff_visc_4 = 0.05;

    // Calculate global thread ID
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (tid < n){
        if (tid==0){
            um_loc_m2 = uL;
            um_loc_m1 = uL;
            um_loc_0 = u[tid];
            um_loc_p1 = u[tid+1];
            um_loc_p2 = u[tid+2];
            }

        else if (tid==1){
            um_loc_m2 = uL;
            um_loc_m1 = u[tid-1];
            um_loc_0 = u[tid];
            um_loc_p1 = u[tid+1];
            um_loc_p2 = u[tid+2];
        }
        else if (tid==n-2){
            um_loc_m2 = u[tid-2];
            um_loc_m1 = u[tid-1];
            um_loc_0 = u[tid];
            um_loc_p1 = u[tid+1];
            um_loc_p2 = uR;
        }
        else if (tid==n-1){
            um_loc_m2 = u[tid-2];
            um_loc_m1 = u[tid-1];
            um_loc_0 = u[tid];
            um_loc_p1 = uR;
            um_loc_p2 = uR;
        }
        else{
            um_loc_m2 = u[tid-2];
            um_loc_m1 = u[tid-1];
            um_loc_0 = u[tid];
            um_loc_p1 = u[tid+1];
            um_loc_p2 = u[tid+2];
        }

        // Left neighbor flux
        um1 = um_loc_m2;
        u0 = um_loc_m1;
        up1 = um_loc_0;
        up2 = um_loc_p1;

        flux_inv_p1 = 0.5 * pow(up1,2);
        flux_inv_0 = 0.5 * pow(u0,2);
        flux_inv = 0.5 * (flux_inv_p1 + flux_inv_0);

        delta_up15 = up2 - up1;
        delta_up05 = up1 - u0;
        delta_um05 = u0 - um1;
        
        flux_vis = coeff_visc_2 * delta_up05 - coeff_visc_4 * (delta_up15 - 2 * delta_up05 + delta_um05);

        h_p05 = flux_inv - flux_vis;

        // Right neighbor flux
        um1 = um_loc_m1;
        u0 = um_loc_0;
        up1 = um_loc_p1;
        up2 = um_loc_p2;

        flux_inv_p1 = 0.5 * pow(up1,2);
        flux_inv_0 = 0.5 * pow(u0,2);
        flux_inv = 0.5 * (flux_inv_p1 + flux_inv_0);


        delta_up15 = up2 - up1;
        delta_up05 = up1 - u0;
        delta_um05 = u0 - um1;
        
        flux_vis = coeff_visc_2 * delta_up05 - coeff_visc_4 * (delta_up15 - 2 * delta_up05 + delta_um05);

        h_m05 = flux_inv - flux_vis;

        // Forward Euler
        dudt[tid] = h_p05 - h_m05;

    }
}

__global__ void update_state(double *__restrict u, const double *__restrict dudt,double dtdx, int n){

    // Calculate global thread ID
    int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    // Flux
    // Boundary check
    if (tid < n) u[tid] = u[tid] + dudt[tid] * dtdx;

    // if (tid == 600) printf("%f",u[tid]);
}

double compute_dt(const std::vector<double>&u, double dx, int n, double CFL_val){

    double dt = dx / abs(2 * u[0]);
    double dt_new;

    for (int i = 0; i < n; i++){
        dt_new = dx / (abs(2 * u[i]) + 1e-8);
        if (dt > dt_new){
            dt = dt_new;
        }
    }

    return dt * CFL_val;
}

int main() {

    // constants
    int n = pow(2, 16);
    // int n = pow(2, 10);
    int bytes = sizeof(double) * n;
    double T = 0.8;

    double xL = -1.0;
    double xR = 1.0;
    double L = xR - xL;
    double dx = L / n;

    double CFL_val = 0.8;


    double x_loc;
    double t=0.0;
    double dt;
    double dtdx;

    // Initialization on CPU
    std::vector<double> u(n, 0.0);
    std::vector<double> dudt(n, 0.0);

    double uL = 1.0;
    double uR = 0.0;
    for (int i = 0; i < n; i++) {
        x_loc = (i + 0.5) * dx + xL;
        if (x_loc < 0.0){
            u[i] = uL;
        }
        else {
            u[i] = uR;
        }
    }

    // Allocate mem on device 
    double *d_u, *d_dudt;
    hipMalloc(&d_u, bytes);
    hipMalloc(&d_dudt, bytes);

    // Copy data from host to the device
    hipMemcpy(d_u, u.data(), bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_dudt, dudt.data(), bytes, hipMemcpyHostToDevice);

    // Threads per CTA (1024)
    int NUM_THREADS = 1 << 10;

    // CTAs per Grid
    // We need to launch at LEAST as many threads as we have elements
    // This equation pads an extra CTA to the grid if N cannot evenly be divided
    // by NUM_THREADS (e.g. N = 1025, NUM_THREADS = 1024)
    int NUM_BLOCKS = (n + NUM_THREADS - 1) / NUM_THREADS;

    // time stepping
    while (t < T){
        // Compute the maximum step size
        // dt = compute_dt(u, dx, n, CFL_val);
        // dt = 0.001 * CFL_val; // HACK
        dt = 1.5 * pow(10.0,-5) * CFL_val; // HACK

        dtdx = dt/dx;

        compute_JST_flux<<<NUM_BLOCKS, NUM_THREADS>>>(d_u, d_dudt, uL, uR, n);

        update_state<<<NUM_BLOCKS, NUM_THREADS>>>(d_u, d_dudt, dtdx, n);

        // Update time
        t = t + dt;
    }

    hipMemcpy(u.data(), d_u, bytes, hipMemcpyDeviceToHost);

    // Create and open an output file stream
    std::ofstream outFile("./output/output_cuda.txt");

    // Check if the file is opened successfully
    if (!outFile) {
        std::cerr << "Error opening file for writing!" << std::endl;
        return 1;
    }

    // Write the array to the file
    for (int i = 0; i < n; i++) {
        outFile << (i + 0.5) * dx + xL << " " << u[i] << "\n";
    }

    // Close the file
    outFile.close();

    std::cout << "Array saved to output.txt in one column" << std::endl;

    return 0;
}